
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <iomanip>

__global__ void kernel(int* c, int* a, int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        c[i] = a[i] + b[i];
    }
}


void print_short(int* a, int size, int border) {
    for (int i = 0; i < border; ++i) {
        std::cout << a[i] << "\t";
    }
    std::cout << "..\t";
    for (int i = size - border; i < size; ++i) {
        std::cout << a[i] << "\t";
    }
    std::cout << std::endl;
}


int main() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(1, 10);

    int size = 512, a[size], b[size], c[size];
    for (int i = 0; i < size; ++i) {
        a[i] = dis(gen);
        b[i] = dis(gen);
    }
    print_short(a, size, 5);
    print_short(b, size, 5);

    int *dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float worktime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    int blocks_opts[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512};

    for (int i = 0; i < 10; ++i) {
        hipEventRecord(start, 0);
        kernel<<<blocks_opts[i], size / blocks_opts[i]>>>(dev_c, dev_a, dev_b, size);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&worktime, start, stop);
        std::cout << blocks_opts[i] << ", " << size / blocks_opts[i] << ":\t" << std::fixed << std::setprecision(16) << worktime << std::endl;
    }

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    print_short(c, size, 5);
    
    return 0;
}